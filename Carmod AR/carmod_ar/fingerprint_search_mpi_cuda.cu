#include <iostream>
#include <fstream>
#include <mpi.h>
#include <hip/hip_runtime.h>

#define FINGERPRINT_SIZE 16
#define MASK_LENGTH 8

void match_fingerprints(unsigned char* d_chunk, unsigned char* d_query, int rows, int* d_result_idx, int* d_result_offset) {
    // Implementation of match_fingerprints function
}

int main(int argc, char** argv) {
    MPI_Init(&argc, &argv);

    int rank, size;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    if (rank == 0) {
        std::cout << "[Master] Loading fingerprint database..." << std::endl;

        std::ifstream infile("fingerprint_database.bin", std::ios::binary);
        if (!infile) {
            std::cerr << "[Master] Cannot open fingerprint_database.bin" << std::endl;
            MPI_Abort(MPI_COMM_WORLD, 1);
        }

        unsigned char* data = new unsigned char[total * FINGERPRINT_SIZE];
        infile.read((char*)data, total * FINGERPRINT_SIZE);
        infile.close();

        std::cout << "[Master] Masked fingerprint to search: ";
        for (int i = 0; i < MASK_LENGTH; i++) std::cout << (int)query[i] << " ";
        std::cout << std::endl;

        // Print first row, first 8 bytes
        std::cout << "[Master] First row, first 8 bytes: ";
        for (int i = 0; i < MASK_LENGTH; ++i) std::cout << (int)data[i] << " ";
        std::cout << std::endl;

        for (int i = 1; i < size; ++i) {
            int start = (i - 1) * chunk_size;
            int count = (i == size - 1) ? total - start : chunk_size;

            MPI_Send(&start, 1, MPI_INT, i, 5, MPI_COMM_WORLD); // Send actual start index
            MPI_Send(query, MASK_LENGTH, MPI_UNSIGNED_CHAR, i, 0, MPI_COMM_WORLD);
            MPI_Send(&count, 1, MPI_INT, i, 1, MPI_COMM_WORLD);
            MPI_Send(data + start * FINGERPRINT_SIZE, count * FINGERPRINT_SIZE, MPI_UNSIGNED_CHAR, i, 2, MPI_COMM_WORLD);
        }
    } else {
        MPI_Status status;
        unsigned char local_query[MASK_LENGTH];
        int local_rows;
        int start_index;

        MPI_Recv(&start_index, 1, MPI_INT, 0, 5, MPI_COMM_WORLD, &status);
        MPI_Recv(local_query, MASK_LENGTH, MPI_UNSIGNED_CHAR, 0, 0, MPI_COMM_WORLD, &status);
        MPI_Recv(&local_rows, 1, MPI_INT, 0, 1, MPI_COMM_WORLD, &status);

        unsigned char* local_data = new unsigned char[local_rows * FINGERPRINT_SIZE];
        MPI_Recv(local_data, local_rows * FINGERPRINT_SIZE, MPI_UNSIGNED_CHAR, 0, 2, MPI_COMM_WORLD, &status);

        // Print received query
        std::cout << "[Worker " << rank << "] Received query: ";
        for (int i = 0; i < MASK_LENGTH; ++i) std::cout << (int)local_query[i] << " ";
        std::cout << std::endl;
        // Print first row, first 8 bytes
        std::cout << "[Worker " << rank << "] First row, first 8 bytes: ";
        for (int i = 0; i < MASK_LENGTH; ++i) std::cout << (int)local_data[i] << " ";
        std::cout << std::endl;

        unsigned char *d_chunk, *d_query;
        int *d_result_idx, *d_result_offset;
        int h_result_idx = -1, h_result_offset = -1;

        hipError_t err;
        err = hipMalloc(&d_chunk, local_rows * FINGERPRINT_SIZE);
        if (err != hipSuccess) { std::cerr << "hipMalloc d_chunk failed: " << hipGetErrorString(err) << std::endl; MPI_Abort(MPI_COMM_WORLD, 1); }
        err = hipMalloc(&d_query, MASK_LENGTH);
        if (err != hipSuccess) { std::cerr << "hipMalloc d_query failed: " << hipGetErrorString(err) << std::endl; MPI_Abort(MPI_COMM_WORLD, 1); }
        err = hipMalloc(&d_result_idx, sizeof(int));
        if (err != hipSuccess) { std::cerr << "hipMalloc d_result_idx failed: " << hipGetErrorString(err) << std::endl; MPI_Abort(MPI_COMM_WORLD, 1); }
        err = hipMalloc(&d_result_offset, sizeof(int));
        if (err != hipSuccess) { std::cerr << "hipMalloc d_result_offset failed: " << hipGetErrorString(err) << std::endl; MPI_Abort(MPI_COMM_WORLD, 1); }

        err = hipMemcpy(d_chunk, local_data, local_rows * FINGERPRINT_SIZE, hipMemcpyHostToDevice);
        if (err != hipSuccess) { std::cerr << "hipMemcpy d_chunk failed: " << hipGetErrorString(err) << std::endl; MPI_Abort(MPI_COMM_WORLD, 1); }
        err = hipMemcpy(d_query, local_query, MASK_LENGTH, hipMemcpyHostToDevice);
        if (err != hipSuccess) { std::cerr << "hipMemcpy d_query failed: " << hipGetErrorString(err) << std::endl; MPI_Abort(MPI_COMM_WORLD, 1); }
        err = hipMemcpy(d_result_idx, &h_result_idx, sizeof(int), hipMemcpyHostToDevice);
        if (err != hipSuccess) { std::cerr << "hipMemcpy d_result_idx failed: " << hipGetErrorString(err) << std::endl; MPI_Abort(MPI_COMM_WORLD, 1); }
        err = hipMemcpy(d_result_offset, &h_result_offset, sizeof(int), hipMemcpyHostToDevice);
        if (err != hipSuccess) { std::cerr << "hipMemcpy d_result_offset failed: " << hipGetErrorString(err) << std::endl; MPI_Abort(MPI_COMM_WORLD, 1); }

        int threads = 256;
        int blocks = (local_rows + threads - 1) / threads;
        match_fingerprints<<<blocks, threads>>>(d_chunk, d_query, local_rows, d_result_idx, d_result_offset);
        err = hipDeviceSynchronize();
        if (err != hipSuccess) { std::cerr << "hipDeviceSynchronize failed: " << hipGetErrorString(err) << std::endl; MPI_Abort(MPI_COMM_WORLD, 1); }

        err = hipMemcpy(&h_result_idx, d_result_idx, sizeof(int), hipMemcpyDeviceToHost);
        if (err != hipSuccess) { std::cerr << "hipMemcpy d_result_idx (to host) failed: " << hipGetErrorString(err) << std::endl; MPI_Abort(MPI_COMM_WORLD, 1); }
        err = hipMemcpy(&h_result_offset, d_result_offset, sizeof(int), hipMemcpyDeviceToHost);
        if (err != hipSuccess) { std::cerr << "hipMemcpy d_result_offset (to host) failed: " << hipGetErrorString(err) << std::endl; MPI_Abort(MPI_COMM_WORLD, 1); }

        if (h_result_idx != -1) {
            std::cout << "[Worker " << rank << "] Match at local index " << h_result_idx
                      << ", offset " << h_result_offset << std::endl;
        } else {
            std::cout << "[Worker " << rank << "] No match found." << std::endl;
        }

        int global_index = (h_result_idx == -1) ? -1 : start_index + h_result_idx;
        MPI_Send(&global_index, 1, MPI_INT, 0, 3, MPI_COMM_WORLD);
        MPI_Send(&h_result_offset, 1, MPI_INT, 0, 4, MPI_COMM_WORLD);

        delete[] local_data;
        hipFree(d_chunk);
        hipFree(d_query);
        hipFree(d_result_idx);
        hipFree(d_result_offset);
    }

    MPI_Finalize();
    return 0;
} 